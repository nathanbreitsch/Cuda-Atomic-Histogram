//question:
//does contention for accessing global memory affect read performance?
//for instance, n threads access each of the n rows in a column at the same time.
//would it be faster for thread i to access row i % n first and proceed to i + j % n?

#define TILE_WIDTH 32
#define DIVIDE_ROUND_UP(a, b)((a + b - 1) / b)
#define GET_INDEX(row, column, numcols)(row * numcols + column)


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>


//define matrix type
typedef struct{
  int row_count;
  int column_count;
  int* elements;
} Matrix;

typedef struct{
  int bin_count;
  int bin_width;
  int* counts;
} Histogram;


Matrix ones(int row_count, int column_count);
Matrix random(int row_count, int column_count);
Histogram make_histogram(Matrix image);
void print_hist(Histogram hist);

void print_matrix(Matrix mat);

int main(){
  //make the matrices you want to multiply
  srand(time(NULL));
  Matrix image = random(512, 512);
  Histogram result = make_histogram(image);
  print_hist(result);

}

//global memory
__global__ void global_atomic_histogram(const Matrix image, Histogram hist){
  int row_index = blockIdx.y * blockDim.y + threadIdx.y;
  int column_index = blockIdx.x * blockDim.x + threadIdx.x;
  int index = row_index * image.column_count + column_index;
  int value = image.elements[index];
  int bin = value / hist.bin_width;
  atomicAdd(&(image.elements[index]), hist.counts[bin]);
  __syncthreads();

}

//shared memory
__global__ void local_atomic_histogram(const Matrix image, Histogram hist){
//todo:
}

Histogram make_histogram(Matrix image){
  hipError_t error;
  //step 1: allocate memory on the kernel for matrix
  Matrix image_d;
  image_d.row_count = image.row_count;
  image_d.column_count = image.column_count;
  size_t image_size = image.row_count * image.column_count * sizeof(int);
  error = hipMalloc((void**) &image_d.elements, image_size);
  if(error != hipSuccess){
    printf("error allocating image matrix\n");
    printf("CUDA error: %s\n", hipGetErrorString(error));
  }
  //step 2: allocate memory on host and device for histogram
  Histogram hist, hist_d;
  hist_d.bin_count = hist.bin_count = 200;
  hist_d.bin_width = hist.bin_width = 1;
  size_t hist_size = hist_d.bin_count * sizeof(int);
  error = hipMalloc((void**) &hist_d.counts, hist_size);
  if(error != hipSuccess){
    printf("error allocating histogram\n");
    printf("CUDA error: %s\n", hipGetErrorString(error));
  }
  hist.counts = (int*) malloc(hist_size);

  //step 3: initialize histogram counts
  for(int i = 0; i < hist.bin_count; i++){
    hist.counts[i] = 0;
  }

  //step 4: copy image to device
  error = hipMemcpy(image_d.elements, image.elements, image_size, hipMemcpyHostToDevice);
  if(error != hipSuccess){ printf("error copying matrix\n"); }
  //step 5: copy histogram zeros do device
  error = hipMemcpy(hist_d.counts, hist.counts, hist_size, hipMemcpyHostToDevice);
  if(error != hipSuccess){ printf("error copying histogram\n"); }

  //step 4: launch kernel

  dim3 block_dims(TILE_WIDTH, TILE_WIDTH);
  dim3 grid_dims(DIVIDE_ROUND_UP(image_d.column_count, block_dims.x), DIVIDE_ROUND_UP(image_d.row_count, block_dims.y));
  global_atomic_histogram <<<grid_dims, block_dims>>> (image_d, hist_d);

  //step 5: copy results back to host
  error = hipMemcpy(hist.counts, hist_d.counts, hist_size, hipMemcpyDeviceToHost);
  if(error != hipSuccess){
  	printf("error copying result histogram\n");
  	printf("CUDA error: %s\n", hipGetErrorString(error));
  }
  return hist;
}

Matrix ones (int row_count, int column_count){
  Matrix result;
  result.row_count = row_count;
  result.column_count = column_count;
  result.elements = (int*) malloc(row_count * column_count * sizeof(int));
  for(int i = 0; i < row_count * column_count; i++){
    result.elements[i] = 1;
  }
  return result;
}

Matrix random (int row_count, int column_count){
  Matrix result;
  result.row_count = row_count;
  result.column_count = column_count;
  result.elements = (int*) malloc(row_count * column_count * sizeof(int));
  for(int i = 0; i < row_count * column_count; i++){
    result.elements[i] = rand() % 200;
  }
  return result;
}

void print_matrix(Matrix mat){
  int num_elements = mat.row_count * mat.column_count;
  for(int i = 0; i < num_elements; i++){
    printf(" %d", mat.elements[i]);
    if(!((i + 1) % mat.column_count)){ printf("\n"); }
  }
}

void print_hist(Histogram hist){
  for(int i = 0; i < hist.bin_count; i++){
    printf(" %d", hist.counts[i]);
  }
}
